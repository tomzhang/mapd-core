#include "hip/hip_runtime.h"
#include <stdint.h>
#include <limits>

extern "C"
__device__ int32_t pos_start_impl() {
  return blockIdx.x * blockDim.x + threadIdx.x;
}

extern "C"
__device__ int32_t pos_step_impl() {
  return blockDim.x * gridDim.x;
}

#define EMPTY_KEY -9223372036854775808L

extern "C" __attribute__((noinline))
__device__ int64_t* get_matching_group_value(int64_t* groups_buffer,
                                  const int32_t h,
                                  const int64_t* key,
                                  const int32_t key_qw_count,
                                  const int32_t agg_col_count) {
  int64_t off = h * (key_qw_count + agg_col_count);
  if (groups_buffer[off] == EMPTY_KEY) {
    memcpy(groups_buffer + off, key, key_qw_count * sizeof(*key));
    return groups_buffer + off + key_qw_count;
  }
  bool match = true;
  for (int64_t i = 0; i < key_qw_count; ++i) {
    if (groups_buffer[off + i] != key[i]) {
      match = false;
      break;
    }
  }
  return match ? groups_buffer + off + key_qw_count : NULL;
}

extern "C" __attribute__((noinline))
__device__ int32_t key_hash(const int64_t* key, const int32_t key_qw_count,
                            const int32_t groups_buffer_entry_count) {
  int32_t hash = 0;
  for (int32_t i = 0; i < key_qw_count; ++i) {
    hash = ((hash << 5) - hash + key[i]) % groups_buffer_entry_count;
  }
  return hash;
}

extern "C" __attribute__((noinline))
__device__ int64_t* get_group_value(int64_t* groups_buffer,
                                    const int32_t groups_buffer_entry_count,
                                    const int64_t* key,
                                    const int32_t key_qw_count,
                                    const int32_t agg_col_count) {
  int64_t h = key_hash(key, key_qw_count, groups_buffer_entry_count);
  int64_t* matching_group = get_matching_group_value(groups_buffer, h, key, key_qw_count, agg_col_count);
  if (matching_group) {
    return matching_group;
  }
  int64_t h_probe = h + 1;
  while (h_probe != h) {
    matching_group = get_matching_group_value(groups_buffer, h_probe, key, key_qw_count, agg_col_count);
    if (matching_group) {
      return matching_group;
    }
    h_probe = (h_probe + 1) % groups_buffer_entry_count;
  }
  // TODO(alex): handle error by resizing?
  return NULL;
}

#define SECSPERMIN	60L
#define MINSPERHOUR	60L
#define HOURSPERDAY	24L
#define SECSPERHOUR	(SECSPERMIN * MINSPERHOUR)
#define SECSPERDAY	(SECSPERHOUR * HOURSPERDAY)
#define DAYSPERWEEK	7
#define MONSPERYEAR	12

#define YEAR_BASE	1900

/* move epoch from 01.01.1970 to 01.03.2000 - this is the first day of new
 * 400-year long cycle, right after additional day of leap year. This adjustment
 * is required only for date calculation, so instead of modifying time_t value
 * (which would require 64-bit operations to work correctly) it's enough to
 * adjust the calculated number of days since epoch. */
#define EPOCH_ADJUSTMENT_DAYS	11017
/* year to which the adjustment was made */
#define ADJUSTED_EPOCH_YEAR	2000
/* 1st March of 2000 is Wednesday */
#define ADJUSTED_EPOCH_WDAY	3
/* there are 97 leap years in 400-year periods. ((400 - 97) * 365 + 97 * 366) */
#define DAYS_PER_400_YEARS	146097L
/* there are 24 leap years in 100-year periods. ((100 - 24) * 365 + 24 * 366) */
#define DAYS_PER_100_YEARS	36524L
/* there is one leap year every 4 years */
#define DAYS_PER_4_YEARS	(3 * 365 + 366)
/* number of days in a non-leap year */
#define DAYS_PER_YEAR		365
/* number of days in January */
#define DAYS_IN_JANUARY		31
/* number of days in non-leap February */
#define DAYS_IN_FEBRUARY	28

extern "C"
__device__  tm* gmtime_r_cuda(const time_t *tim_p, tm* res) {
  const int month_lengths[2][MONSPERYEAR] = {
    {31, 28, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31},
    {31, 29, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31}
  };
  long days, rem;
  const time_t lcltime = *tim_p;
  int year, month, yearday, weekday;
  int years400, years100, years4, remainingyears;
  int yearleap;
  const int *ip;

  days = ((long)lcltime) / SECSPERDAY - EPOCH_ADJUSTMENT_DAYS;
  rem = ((long)lcltime) % SECSPERDAY;
  if (rem < 0)
    {
      rem += SECSPERDAY;
      --days;
    }

  /* compute hour, min, and sec */
  res->tm_hour = (int) (rem / SECSPERHOUR);
  rem %= SECSPERHOUR;
  res->tm_min = (int) (rem / SECSPERMIN);
  res->tm_sec = (int) (rem % SECSPERMIN);

  /* compute day of week */
  if ((weekday = ((ADJUSTED_EPOCH_WDAY + days) % DAYSPERWEEK)) < 0)
    weekday += DAYSPERWEEK;
  res->tm_wday = weekday;

  /* compute year & day of year */
  years400 = days / DAYS_PER_400_YEARS;
  days -= years400 * DAYS_PER_400_YEARS;
  /* simplify by making the values positive */
  if (days < 0)
    {
      days += DAYS_PER_400_YEARS;
      --years400;
    }

  years100 = days / DAYS_PER_100_YEARS;
  if (years100 == 4) /* required for proper day of year calculation */
    --years100;
  days -= years100 * DAYS_PER_100_YEARS;
  years4 = days / DAYS_PER_4_YEARS;
  days -= years4 * DAYS_PER_4_YEARS;
  remainingyears = days / DAYS_PER_YEAR;
  if (remainingyears == 4) /* required for proper day of year calculation */
    --remainingyears;
  days -= remainingyears * DAYS_PER_YEAR;

  year = ADJUSTED_EPOCH_YEAR + years400 * 400 + years100 * 100 + years4 * 4 +
      remainingyears;

  /* If remainingyears is zero, it means that the years were completely
   * "consumed" by modulo calculations by 400, 100 and 4, so the year is:
   * 1. a multiple of 4, but not a multiple of 100 or 400 - it's a leap year,
   * 2. a multiple of 4 and 100, but not a multiple of 400 - it's not a leap
   * year,
   * 3. a multiple of 4, 100 and 400 - it's a leap year.
   * If years4 is non-zero, it means that the year is not a multiple of 100 or
   * 400 (case 1), so it's a leap year. If years100 is zero (and years4 is zero
   * - due to short-circuiting), it means that the year is a multiple of 400
   * (case 3), so it's also a leap year. */
  yearleap = remainingyears == 0 && (years4 != 0 || years100 == 0);

  /* adjust back to 1st January */
  yearday = days + DAYS_IN_JANUARY + DAYS_IN_FEBRUARY + yearleap;
  if (yearday >= DAYS_PER_YEAR + yearleap)
    {
      yearday -= DAYS_PER_YEAR + yearleap;
      ++year;
    }
  res->tm_yday = yearday;
  res->tm_year = year - YEAR_BASE;

  /* Because "days" is the number of days since 1st March, the additional leap
   * day (29th of February) is the last possible day, so it doesn't matter much
   * whether the year is actually leap or not. */
  ip = month_lengths[1];
  month = 2;
  while (days >= ip[month])
    {
      days -= ip[month];
      if (++month >= MONSPERYEAR)
        month = 0;
    }
  res->tm_mon = month;
  res->tm_mday = days + 1;

  res->tm_isdst = 0;

  return (res);
}

#include "ExtractFromTime.cpp"
