#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <limits>
#include "GpuRtConstants.h"

extern "C" __device__ int32_t pos_start_impl(const int32_t* row_index_resume) {
  return blockIdx.x * blockDim.x + threadIdx.x + (row_index_resume ? row_index_resume[blockIdx.x] : 0);
}

extern "C" __device__ int32_t group_buff_idx_impl() {
  return pos_start_impl(NULL);
}

extern "C" __device__ int32_t pos_step_impl() {
  return blockDim.x * gridDim.x;
}

extern "C" __device__ int8_t thread_warp_idx(const int8_t warp_sz) {
  return threadIdx.x % warp_sz;
}

extern "C" __device__ const int64_t* init_shared_mem_nop(const int64_t* groups_buffer,
                                                         const int32_t groups_buffer_size) {
  return groups_buffer;
}

extern "C" __device__ void write_back_nop(int64_t* dest, int64_t* src, const int32_t sz) {
}

extern "C" __device__ const int64_t* init_shared_mem(const int64_t* groups_buffer, const int32_t groups_buffer_size) {
  extern __shared__ int64_t fast_bins[];
  if (threadIdx.x == 0) {
    memcpy(fast_bins, groups_buffer, groups_buffer_size);
  }
  __syncthreads();
  return fast_bins;
}

extern "C" __device__ void write_back(int64_t* dest, int64_t* src, const int32_t sz) {
  __syncthreads();
  if (threadIdx.x == 0) {
    memcpy(dest, src, sz);
  }
}

#define init_group_by_buffer_gpu_impl init_group_by_buffer_gpu

#include "GpuInitGroups.cu"

#undef init_group_by_buffer_gpu_impl

extern "C" __device__ int64_t* get_matching_group_value(int64_t* groups_buffer,
                                                        const uint32_t h,
                                                        const int64_t* key,
                                                        const uint32_t key_qw_count,
                                                        const uint32_t agg_col_count,
                                                        const int64_t* init_vals) {
  uint32_t off = h * (key_qw_count + agg_col_count);
  {
    const uint64_t old = atomicCAS(reinterpret_cast<unsigned long long*>(groups_buffer + off), EMPTY_KEY, *key);
    if (EMPTY_KEY == old) {
      memcpy(groups_buffer + off, key, key_qw_count * sizeof(*key));
      memcpy(groups_buffer + off + key_qw_count, init_vals, agg_col_count * sizeof(*init_vals));
    }
  }
  __syncthreads();
  bool match = true;
  for (uint32_t i = 0; i < key_qw_count; ++i) {
    if (groups_buffer[off + i] != key[i]) {
      match = false;
      break;
    }
  }
  return match ? groups_buffer + off + key_qw_count : NULL;
}

#include "GroupByRuntime.cpp"

__device__ int64_t atomicMax64(int64_t* address, int64_t val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, max((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

__device__ int64_t atomicMin64(int64_t* address, int64_t val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, min((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

// As of 20160418, CUDA 8.0EA only defines `atomicAdd(double*, double)` for compute capability >= 6.0.
#if CUDA_VERSION < 8000 || (defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600)
__device__ double atomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

__device__ double atomicMax(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(max(val, __longlong_as_double(assumed))));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

__device__ double atomicMin(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(min(val, __longlong_as_double(assumed))));
  } while (assumed != old);

  return __longlong_as_double(old);
}

extern "C" __device__ void agg_count_shared(int64_t* agg, const int64_t val) {
  atomicAdd(reinterpret_cast<int32_t*>(agg), 1L);
}

extern "C" __device__ void agg_count_double_shared(int64_t* agg, const double val) {
  agg_count_shared(agg, val);
}

extern "C" __device__ void agg_sum_shared(int64_t* agg, const int64_t val) {
  atomicAdd(reinterpret_cast<unsigned long long*>(agg), val);
}

extern "C" __device__ void agg_sum_double_shared(int64_t* agg, const double val) {
  atomicAdd(reinterpret_cast<double*>(agg), val);
}

extern "C" __device__ void agg_max_shared(int64_t* agg, const int64_t val) {
  atomicMax64(agg, val);
}

extern "C" __device__ void agg_max_double_shared(int64_t* agg, const double val) {
  atomicMax(reinterpret_cast<double*>(agg), val);
}

extern "C" __device__ void agg_min_shared(int64_t* agg, const int64_t val) {
  atomicMin64(agg, val);
}

extern "C" __device__ void agg_min_double_shared(int64_t* agg, const double val) {
  atomicMin(reinterpret_cast<double*>(agg), val);
}

extern "C" __device__ void agg_id_shared(int64_t* agg, const int64_t val) {
  *agg = val;
}

extern "C" __device__ void agg_id_double_shared(int64_t* agg, const double val) {
  *agg = *(reinterpret_cast<const int64_t*>(&val));
}

#define DEF_SKIP_AGG(base_agg_func)                              \
  extern "C" __device__ void base_agg_func##_skip_val_shared(    \
      int64_t* agg, const int64_t val, const int64_t skip_val) { \
    if (val != skip_val) {                                       \
      base_agg_func##_shared(agg, val);                          \
    }                                                            \
  }

DEF_SKIP_AGG(agg_count)
DEF_SKIP_AGG(agg_sum)

__device__ int64_t atomicMin64SkipVal(int64_t* address, int64_t val, const int64_t skip_val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, assumed == skip_val ? val : min((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

extern "C" __device__ void agg_min_skip_val_shared(int64_t* agg, const int64_t val, const int64_t skip_val) {
  if (val != skip_val) {
    atomicMin64SkipVal(agg, val, skip_val);
  }
}

__device__ int64_t atomicMax64SkipVal(int64_t* address, int64_t val, const int64_t skip_val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, assumed == skip_val ? val : max((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

extern "C" __device__ void agg_max_skip_val_shared(int64_t* agg, const int64_t val, const int64_t skip_val) {
  if (val != skip_val) {
    atomicMax64SkipVal(agg, val, skip_val);
  }
}

#undef DEF_SKIP_AGG

#define DEF_SKIP_AGG(base_agg_func)                                                                                   \
  extern "C" __device__ void base_agg_func##_skip_val_shared(int64_t* agg, const double val, const double skip_val) { \
    if (val != skip_val) {                                                                                            \
      base_agg_func##_shared(agg, val);                                                                               \
    }                                                                                                                 \
  }

DEF_SKIP_AGG(agg_count_double)
DEF_SKIP_AGG(agg_sum_double)

__device__ double atomicMinDblSkipVal(double* address, double val, const double skip_val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull;
  unsigned long long int skip_val_as_ull = *((unsigned long long int*)&skip_val);
  unsigned long long int assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull,
                    assumed,
                    assumed == skip_val_as_ull ? *((unsigned long long int*)&val)
                                               : __double_as_longlong(min(val, __longlong_as_double(assumed))));
  } while (assumed != old);

  return __longlong_as_double(old);
}

extern "C" __device__ void agg_min_double_skip_val_shared(int64_t* agg, const double val, const double skip_val) {
  if (val != skip_val) {
    atomicMinDblSkipVal(reinterpret_cast<double*>(agg), val, skip_val);
  }
}

__device__ double atomicMaxDblSkipVal(double* address, double val, const double skip_val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull;
  unsigned long long int skip_val_as_ull = *((unsigned long long int*)&skip_val);
  unsigned long long int assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull,
                    assumed,
                    assumed == skip_val_as_ull ? *((unsigned long long int*)&val)
                                               : __double_as_longlong(max(val, __longlong_as_double(assumed))));
  } while (assumed != old);

  return __longlong_as_double(old);
}

extern "C" __device__ void agg_max_double_skip_val_shared(int64_t* agg, const double val, const double skip_val) {
  if (val != skip_val) {
    atomicMaxDblSkipVal(reinterpret_cast<double*>(agg), val, skip_val);
  }
}

#undef DEF_SKIP_AGG

#include "ExtractFromTime.cpp"
#include "DateTruncate.cpp"
#include "../Utils/ChunkIter.cpp"
#define EXECUTE_INCLUDE
#include "ArrayOps.cpp"
#include "StringFunctions.cpp"
#undef EXECUTE_INCLUDE
#include "../Utils/StringLike.cpp"

extern "C" __device__ uint64_t string_decode(int8_t* chunk_iter_, int64_t pos) {
  // TODO(alex): de-dup, the x64 version is basically identical
  ChunkIter* chunk_iter = reinterpret_cast<ChunkIter*>(chunk_iter_);
  VarlenDatum vd;
  bool is_end;
  ChunkIter_get_nth(chunk_iter, pos, false, &vd, &is_end);
  return vd.is_null ? 0 : (reinterpret_cast<uint64_t>(vd.pointer) & 0xffffffffffff) |
                              (static_cast<uint64_t>(vd.length) << 48);
}

extern "C" __device__ int32_t merge_error_code(const int32_t err_code, int32_t* merged_err_code) {
  if (err_code) {
    int32_t assumed = *merged_err_code;
    int32_t old;
    do {
      old = atomicCAS(merged_err_code, assumed, err_code);
    } while (old != assumed);
  }
  __syncthreads();
  return *merged_err_code;
}
