#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <limits>
#include "ExtensionFunctions.hpp"
#include "GpuRtConstants.h"

extern "C" __device__ int32_t pos_start_impl(const int32_t* row_index_resume) {
  return blockIdx.x * blockDim.x + threadIdx.x;
}

extern "C" __device__ int32_t group_buff_idx_impl() {
  return pos_start_impl(NULL);
}

extern "C" __device__ int32_t pos_step_impl() {
  return blockDim.x * gridDim.x;
}

extern "C" __device__ int8_t thread_warp_idx(const int8_t warp_sz) {
  return threadIdx.x % warp_sz;
}

extern "C" __device__ const int64_t* init_shared_mem_nop(const int64_t* groups_buffer,
                                                         const int32_t groups_buffer_size) {
  return groups_buffer;
}

extern "C" __device__ void write_back_nop(int64_t* dest, int64_t* src, const int32_t sz) {
}

extern "C" __device__ const int64_t* init_shared_mem(const int64_t* groups_buffer, const int32_t groups_buffer_size) {
  extern __shared__ int64_t fast_bins[];
  if (threadIdx.x == 0) {
    memcpy(fast_bins, groups_buffer, groups_buffer_size);
  }
  __syncthreads();
  return fast_bins;
}

extern "C" __device__ void write_back(int64_t* dest, int64_t* src, const int32_t sz) {
  __syncthreads();
  if (threadIdx.x == 0) {
    memcpy(dest, src, sz);
  }
}

#define init_group_by_buffer_gpu_impl init_group_by_buffer_gpu

#include "GpuInitGroups.cu"

#undef init_group_by_buffer_gpu_impl

extern "C" __device__ int64_t* get_matching_group_value(int64_t* groups_buffer,
                                                        const uint32_t h,
                                                        const int64_t* key,
                                                        const uint32_t key_qw_count,
                                                        const uint32_t row_size_quad,
                                                        const int64_t* init_vals) {
  uint32_t off = h * row_size_quad;
  {
    const uint64_t old = atomicCAS(reinterpret_cast<unsigned long long*>(groups_buffer + off), EMPTY_KEY_64, *key);
    if (EMPTY_KEY_64 == old) {
      memcpy(groups_buffer + off, key, key_qw_count * sizeof(int64_t));
      memcpy(groups_buffer + off + key_qw_count, init_vals, (row_size_quad - key_qw_count) * sizeof(int64_t));
    }
  }
  __syncthreads();
  bool match = true;
  for (uint32_t i = 0; i < key_qw_count; ++i) {
    if (groups_buffer[off + i] != key[i]) {
      match = false;
      break;
    }
  }
  return match ? groups_buffer + off + key_qw_count : NULL;
}

extern "C" __device__ int64_t* get_matching_group_value_columnar(int64_t* groups_buffer,
                                                                 const uint32_t h,
                                                                 const int64_t* key,
                                                                 const uint32_t key_qw_count,
                                                                 const size_t entry_count) {
  uint32_t off = h;
  {
    const uint64_t old = atomicCAS(reinterpret_cast<unsigned long long*>(groups_buffer + off), EMPTY_KEY_64, *key);
    if (EMPTY_KEY_64 == old) {
      for (size_t i = 0; i < key_qw_count; ++i) {
        groups_buffer[off] = key[i];
        off += entry_count;
      }
      return &groups_buffer[off];
    }
  }
  __syncthreads();
  off = h;
  for (size_t i = 0; i < key_qw_count; ++i) {
    if (groups_buffer[off] != key[i]) {
      return NULL;
    }
    off += entry_count;
  }
  return &groups_buffer[off];
}

#include "GroupByRuntime.cpp"

__device__ int64_t atomicMax64(int64_t* address, int64_t val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, max((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

__device__ int64_t atomicMin64(int64_t* address, int64_t val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, min((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

// As of 20160418, CUDA 8.0EA only defines `atomicAdd(double*, double)` for compute capability >= 6.0.
#if CUDA_VERSION < 8000 || (defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600)
__device__ double atomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

__device__ double atomicMax(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(max(val, __longlong_as_double(assumed))));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

__device__ float atomicMax(float* address, float val) {
  int* address_as_int = (int*)address;
  int old = *address_as_int, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_int, assumed, __float_as_int(max(val, __int_as_float(assumed))));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __int_as_float(old);
}

__device__ double atomicMin(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(min(val, __longlong_as_double(assumed))));
  } while (assumed != old);

  return __longlong_as_double(old);
}

__device__ double atomicMin(float* address, float val) {
  int* address_as_ull = (int*)address;
  int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __float_as_int(min(val, __int_as_float(assumed))));
  } while (assumed != old);

  return __int_as_float(old);
}

extern "C" __device__ uint64_t agg_count_shared(uint64_t* agg, const int64_t val) {
  return static_cast<uint64_t>(atomicAdd(reinterpret_cast<uint32_t*>(agg), 1UL));
}

extern "C" __device__ uint32_t agg_count_int32_shared(uint32_t* agg, const int32_t val) {
  return atomicAdd(agg, 1UL);
}

extern "C" __device__ uint64_t agg_count_double_shared(uint64_t* agg, const double val) {
  return agg_count_shared(agg, val);
}

extern "C" __device__ uint32_t agg_count_float_shared(uint32_t* agg, const float val) {
  return agg_count_int32_shared(agg, val);
}

extern "C" __device__ int64_t agg_sum_shared(int64_t* agg, const int64_t val) {
  return atomicAdd(reinterpret_cast<unsigned long long*>(agg), val);
}

extern "C" __device__ int32_t agg_sum_int32_shared(int32_t* agg, const int32_t val) {
  return atomicAdd(agg, val);
}

extern "C" __device__ void agg_sum_float_shared(int32_t* agg, const float val) {
  atomicAdd(reinterpret_cast<float*>(agg), val);
}

extern "C" __device__ void agg_sum_double_shared(int64_t* agg, const double val) {
  atomicAdd(reinterpret_cast<double*>(agg), val);
}

extern "C" __device__ void agg_max_shared(int64_t* agg, const int64_t val) {
  atomicMax64(agg, val);
}

extern "C" __device__ void agg_max_int32_shared(int32_t* agg, const int32_t val) {
  atomicMax(agg, val);
}

extern "C" __device__ void agg_max_double_shared(int64_t* agg, const double val) {
  atomicMax(reinterpret_cast<double*>(agg), val);
}

extern "C" __device__ void agg_max_float_shared(int32_t* agg, const float val) {
  atomicMax(reinterpret_cast<float*>(agg), val);
}

extern "C" __device__ void agg_min_shared(int64_t* agg, const int64_t val) {
  atomicMin64(agg, val);
}

extern "C" __device__ void agg_min_int32_shared(int32_t* agg, const int32_t val) {
  atomicMin(agg, val);
}

extern "C" __device__ void agg_min_double_shared(int64_t* agg, const double val) {
  atomicMin(reinterpret_cast<double*>(agg), val);
}

extern "C" __device__ void agg_min_float_shared(int32_t* agg, const float val) {
  atomicMin(reinterpret_cast<float*>(agg), val);
}

extern "C" __device__ void agg_id_shared(int64_t* agg, const int64_t val) {
  *agg = val;
}

extern "C" __device__ void agg_id_int32_shared(int32_t* agg, const int32_t val) {
  *agg = val;
}

extern "C" __device__ void agg_id_double_shared(int64_t* agg, const double val) {
  *agg = *(reinterpret_cast<const int64_t*>(&val));
}

extern "C" __device__ void agg_id_double_shared_slow(int64_t* agg, const double* val) {
  *agg = *(reinterpret_cast<const int64_t*>(val));
}

extern "C" __device__ void agg_id_float_shared(int32_t* agg, const float val) {
  *agg = __float_as_int(val);
}

#define DEF_SKIP_AGG(base_agg_func)                                                                                    \
  extern "C" __device__ ADDR_T base_agg_func##_skip_val_shared(ADDR_T* agg, const DATA_T val, const DATA_T skip_val) { \
    if (val != skip_val) {                                                                                             \
      return base_agg_func##_shared(agg, val);                                                                         \
    }                                                                                                                  \
    return 0;                                                                                                          \
  }

#define DATA_T int64_t
#define ADDR_T uint64_t
DEF_SKIP_AGG(agg_count)
#undef DATA_T
#undef ADDR_T

#define DATA_T int32_t
#define ADDR_T uint32_t
DEF_SKIP_AGG(agg_count_int32)
#undef DATA_T
#undef ADDR_T

// Initial value for nullable column is INT32_MIN
extern "C" __device__ void agg_max_int32_skip_val_shared(int32_t* agg, const int32_t val, const int32_t skip_val) {
  if (val != skip_val) {
    agg_max_int32_shared(agg, val);
  }
}

__device__ int32_t atomicMin32SkipVal(int32_t* address, int32_t val, const int32_t skip_val) {
  int32_t old = *address, assumed;

  do {
    assumed = old;
    old = atomicCAS(address, assumed, assumed == skip_val ? val : min(val, assumed));
  } while (assumed != old);

  return old;
}

extern "C" __device__ void agg_min_int32_skip_val_shared(int32_t* agg, const int32_t val, const int32_t skip_val) {
  if (val != skip_val) {
    atomicMin32SkipVal(agg, val, skip_val);
  }
}

__device__ int32_t atomicSum32SkipVal(int32_t* address, const int32_t val, const int32_t skip_val) {
  unsigned int* address_as_int = (unsigned int*)address;
  int32_t old = atomicExch(address_as_int, 0);
  int32_t old2 = atomicAdd(address_as_int, old == skip_val ? val : (val + old));
  return old == skip_val ? old2 : (old2 + old);
}

extern "C" __device__ int32_t agg_sum_int32_skip_val_shared(int32_t* agg, const int32_t val, const int32_t skip_val) {
  if (val != skip_val) {
    const int32_t old = atomicSum32SkipVal(agg, val, skip_val);
    return old;
  }
  return 0;
}

__device__ int64_t atomicSum64SkipVal(int64_t* address, const int64_t val, const int64_t skip_val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  int64_t old = atomicExch(address_as_ull, 0);
  int64_t old2 = atomicAdd(address_as_ull, old == skip_val ? val : (val + old));
  return old == skip_val ? old2 : (old2 + old);
}

extern "C" __device__ int64_t agg_sum_skip_val_shared(int64_t* agg, const int64_t val, const int64_t skip_val) {
  if (val != skip_val) {
    return atomicSum64SkipVal(agg, val, skip_val);
  }
  return 0;
}

__device__ int64_t atomicMin64SkipVal(int64_t* address, int64_t val, const int64_t skip_val) {
  unsigned long long int* address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, assumed == skip_val ? val : min((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

extern "C" __device__ void agg_min_skip_val_shared(int64_t* agg, const int64_t val, const int64_t skip_val) {
  if (val != skip_val) {
    atomicMin64SkipVal(agg, val, skip_val);
  }
}

__device__ int64_t atomicMax64SkipVal(int64_t* address, int64_t val, const int64_t skip_val) {
  unsigned long long int* address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, assumed == skip_val ? val : max((long long)val, (long long)assumed));
  } while (assumed != old);

  return old;
}

extern "C" __device__ void agg_max_skip_val_shared(int64_t* agg, const int64_t val, const int64_t skip_val) {
  if (val != skip_val) {
    atomicMax64SkipVal(agg, val, skip_val);
  }
}

#undef DEF_SKIP_AGG
#define DEF_SKIP_AGG(base_agg_func)                                                                                    \
  extern "C" __device__ ADDR_T base_agg_func##_skip_val_shared(ADDR_T* agg, const DATA_T val, const DATA_T skip_val) { \
    if (val != skip_val) {                                                                                             \
      return base_agg_func##_shared(agg, val);                                                                         \
    }                                                                                                                  \
    return *agg;                                                                                                       \
  }

#define DATA_T double
#define ADDR_T uint64_t
DEF_SKIP_AGG(agg_count_double)
#undef ADDR_T
#undef DATA_T

#define DATA_T float
#define ADDR_T uint32_t
DEF_SKIP_AGG(agg_count_float)
#undef ADDR_T
#undef DATA_T

// Initial value for nullable column is FLOAT_MIN
extern "C" __device__ void agg_max_float_skip_val_shared(int32_t* agg, const float val, const float skip_val) {
  if (val != skip_val) {
    agg_max_float_shared(agg, val);
  }
}

__device__ double atomicMinFltSkipVal(int32_t* address, float val, const float skip_val) {
  int32_t old = *address;
  int32_t skip_val_as_int = __float_as_int(skip_val);
  int32_t assumed;

  do {
    assumed = old;
    old =
        atomicCAS(address,
                  assumed,
                  assumed == skip_val_as_int ? __float_as_int(val) : __float_as_int(min(val, __int_as_float(assumed))));
  } while (assumed != old);

  return __float_as_int(old);
}

extern "C" __device__ void agg_min_float_skip_val_shared(int32_t* agg, const float val, const float skip_val) {
  if (val != skip_val) {
    atomicMinFltSkipVal(agg, val, skip_val);
  }
}

__device__ void atomicSumFltSkipVal(float* address, const float val, const float skip_val) {
  unsigned int* address_as_int = (unsigned*)address;
  int32_t old = atomicExch(address_as_int, __float_as_int(0.));
  atomicAdd(address_as_int, __float_as_int(old == __float_as_int(skip_val) ? val : (val + __int_as_float(old))));
}

extern "C" __device__ void agg_sum_float_skip_val_shared(int32_t* agg, const float val, const float skip_val) {
  if (val != skip_val) {
    atomicSumFltSkipVal(reinterpret_cast<float*>(agg), val, skip_val);
  }
}

__device__ void atomicSumDblSkipVal(double* address, const double val, const double skip_val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  double old = __longlong_as_double(atomicExch(address_as_ull, __double_as_longlong(0.)));
  atomicAdd(address_as_ull, __double_as_longlong(old == skip_val ? val : (val + old)));
}

extern "C" __device__ void agg_sum_double_skip_val_shared(int64_t* agg, const double val, const double skip_val) {
  if (val != skip_val) {
    atomicSumDblSkipVal(reinterpret_cast<double*>(agg), val, skip_val);
  }
}

__device__ double atomicMinDblSkipVal(double* address, double val, const double skip_val) {
  unsigned long long int* address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull;
  unsigned long long int skip_val_as_ull = *reinterpret_cast<const unsigned long long*>(&skip_val);
  unsigned long long int assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull,
                    assumed,
                    assumed == skip_val_as_ull ? *reinterpret_cast<unsigned long long*>(&val)
                                               : __double_as_longlong(min(val, __longlong_as_double(assumed))));
  } while (assumed != old);

  return __longlong_as_double(old);
}

extern "C" __device__ void agg_min_double_skip_val_shared(int64_t* agg, const double val, const double skip_val) {
  if (val != skip_val) {
    atomicMinDblSkipVal(reinterpret_cast<double*>(agg), val, skip_val);
  }
}

__device__ double atomicMaxDblSkipVal(double* address, double val, const double skip_val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull;
  unsigned long long int skip_val_as_ull = *((unsigned long long int*)&skip_val);
  unsigned long long int assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull,
                    assumed,
                    assumed == skip_val_as_ull ? *((unsigned long long int*)&val)
                                               : __double_as_longlong(max(val, __longlong_as_double(assumed))));
  } while (assumed != old);

  return __longlong_as_double(old);
}

extern "C" __device__ void agg_max_double_skip_val_shared(int64_t* agg, const double val, const double skip_val) {
  if (val != skip_val) {
    atomicMaxDblSkipVal(reinterpret_cast<double*>(agg), val, skip_val);
  }
}

#undef DEF_SKIP_AGG

#include "ExtractFromTime.cpp"
#include "DateTruncate.cpp"
#include "../Utils/ChunkIter.cpp"
#define EXECUTE_INCLUDE
#include "ArrayOps.cpp"
#include "StringFunctions.cpp"
#undef EXECUTE_INCLUDE
#include "../Utils/StringLike.cpp"

extern "C" __device__ uint64_t string_decode(int8_t* chunk_iter_, int64_t pos) {
  // TODO(alex): de-dup, the x64 version is basically identical
  ChunkIter* chunk_iter = reinterpret_cast<ChunkIter*>(chunk_iter_);
  VarlenDatum vd;
  bool is_end;
  ChunkIter_get_nth(chunk_iter, pos, false, &vd, &is_end);
  return vd.is_null ? 0 : (reinterpret_cast<uint64_t>(vd.pointer) & 0xffffffffffff) |
                              (static_cast<uint64_t>(vd.length) << 48);
}

extern "C" __device__ void force_sync() {
  __threadfence_block();
}
